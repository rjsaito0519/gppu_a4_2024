#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <chrono>
#include <iostream>
#include <TVector3.h>
#include "tracking_cuda.h"

__global__ void houghTransformKernel(int *hough_space, const float *x_data, const float *z_data, int data_size, int n_rho) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < data_size) {
        float x = x_data[index];
        float z = z_data[index];

        for (int theta = 0; theta <= 180; ++theta) {
            float radian = theta * M_PI / 180.0;
            int rho = static_cast<int>(round(z * cosf(radian) + x * sinf(radian)) + (n_rho - 1) / 2);
            atomicAdd(&hough_space[theta * n_rho + rho], 1);
        }
    }
}

std::vector<std::vector<int>> tracking_cuda(const std::vector<TVector3>& pos_container) {
    int max_iter = pos_container.size();
    std::vector<int> track_id_container(max_iter, -1);
    std::vector<std::vector<int>> indices(10);
    int track_id = 0;

    while (std::count(track_id_container.begin(), track_id_container.end(), -1) > 5 && track_id < 10) {
        auto start_time = std::chrono::high_resolution_clock::now();

        std::vector<float> host_x_data, host_z_data;
        float most_far_position = 0.0;
        for (int i = 0; i < max_iter; i++) {
            if (track_id_container[i] == -1) {
                host_x_data.push_back(pos_container[i].X());
                host_z_data.push_back(pos_container[i].Z());
                if (std::abs(pos_container[i].X()) > most_far_position || std::abs(pos_container[i].Z()) > most_far_position) {
                    most_far_position = std::max(std::abs(pos_container[i].X()), std::abs(pos_container[i].Z()));
                }
            }
        }

        int data_size = host_x_data.size();
        float *cuda_x_data, *cuda_z_data;
        int *cuda_hough_space;
        int n_rho = 2 * static_cast<int>(std::ceil(most_far_position * std::sqrt(2.0))) + 1;

        hipMalloc(&cuda_x_data, data_size * sizeof(float));
        hipMalloc(&cuda_z_data, data_size * sizeof(float));
        hipMalloc(&cuda_hough_space, 181 * n_rho * sizeof(int));

        hipMemcpy(cuda_x_data, host_x_data.data(), data_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(cuda_z_data, host_z_data.data(), data_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemset(cuda_hough_space, 0, 181 * n_rho * sizeof(int));

        int threadsPerBlock = 256;
        int blocksPerGrid = (data_size + threadsPerBlock - 1) / threadsPerBlock;

        houghTransformKernel<<<blocksPerGrid, threadsPerBlock>>>(cuda_hough_space, cuda_x_data, cuda_z_data, data_size, n_rho);

        std::vector<int> host_hough_space(181 * n_rho);
        hipMemcpy(host_hough_space.data(), cuda_hough_space, 181 * n_rho * sizeof(int), hipMemcpyDeviceToHost);

        hipFree(cuda_x_data);
        hipFree(cuda_z_data);
        hipFree(cuda_hough_space);

        auto max_it = std::max_element(host_hough_space.begin(), host_hough_space.end());
        int max_index = std::distance(host_hough_space.begin(), max_it);
        int max_theta = max_index / n_rho;
        int max_rho = max_index % n_rho - static_cast<int>((n_rho - 1) / 2);

        double bin_diff;
        int max_diff = 4;
        for (int i = 0; i < max_iter; i++) {
            if (track_id_container[i] != -1) continue;
            bool within_circ = false;
            for (int theta = max_theta - max_diff; theta <= max_theta + max_diff; theta++) {
                double rho = std::cos(theta * M_PI / 180.0) * pos_container[i].Z() + std::sin(theta * M_PI / 180.0) * pos_container[i].X();
                double diff = std::abs(max_rho - rho) + std::abs(max_theta - theta);
                if (diff < max_diff) within_circ = true;
            }
            if (within_circ) {
                track_id_container[i] = track_id;
                indices[track_id].push_back(i);
            }
        }
        track_id++;
    }

    return indices;
}
